#include <algorithm>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void add(int n, float* x, float*y, float* z) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx<n)
    z[idx] = x[idx] + y[idx];
}

int main(void) {
  int N = 1<<20;

  float* x = new float[N];
  float* y = new float[N];
  float* z = new float[N];

  for (int i=0; i<N; i++){
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  float *d_x, *d_y, *d_z;

  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));
  hipMalloc(&d_z, N * sizeof(float));

  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  
  // Create CUDA events to measure time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record the start event
  hipEventRecord(start);
  
  add<<<numBlocks, blockSize>>>(N, d_x, d_y, d_z);
  
  // Record the stop event
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Calculate the time elapsed between the two events
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  cout << "Kernel execution time: " << milliseconds << " milliseconds" << endl;
  
  hipMemcpy(z, d_z, N * sizeof(float), hipMemcpyDeviceToHost);
  
  float err = 0.0;
  for (int i=0; i<N; i++){
    err += fabs(z[i] - 3.0f);
  }
  cout << "error is" << err << endl;

  // free device memory
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  // free host memory
  delete[] x;
  delete[] y;
  delete[] z;
  
  return 0;
}
